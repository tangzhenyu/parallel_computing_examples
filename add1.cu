
#include <hip/hip_runtime.h>
#include <stdio.h>

//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

#define N 1000

//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//
__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int main() {
    //
    // Create int arrays on the CPU.
    // ('h' stands for "host".)
    //
    int ha[N], hb[N];

    //
    // Create corresponding int arrays on the GPU.
    // ('d' stands for "device".)
    //
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    //
    // Initialise the input data on the CPU.
    //
    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }

    //
    // Copy input data to array on GPU.
    //
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    //
    // Launch GPU code with N threads, one per
    // array element.
    //
    add<<<N, 1>>>(da, db);

    //
    // Copy output array from GPU back to CPU.
    //
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    //
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);

    return 0;
}
